#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include "MDArrayHelper.h"
#include "MDUtils.h"
#include "MDStencil.h"

using namespace std;


/*
Note: The kernel may need shared memory optimization
*/

/*
GPU kernel to perform dim dimentional stencil 
on a data including boundary data

in: input array for input data including boundary
out: output array for output data including boundary unchanged
arraySize: size of in and out for each dimention
wArr: weight array
wArrSize: size of wArr for each dimention

NOTE: Size of the data part of in and out (without boundaries) is a multiple of block side (number of thread per block)^(1/dim)
*/
__global__
void stencilKernel (float *in, float *out, int *arrSize, float *wArr, int *wArrSize, int dim, int blockSide)
{
    // create index related local variables
    int *blockSize = new int[dim];
    for (int i=0; i<dim; i++) blockSize[i] = blockSide;

    int *radius = new int[dim];
    for (int i=0; i<dim; i++) radius[i] = wArrSize[i] / 2;

    int *dataSize = new int[dim];
    for (int i=0; i<dim; i++) dataSize[i] = arrSize[i] - 2 * radius[i];

    int *gridSize = new int[dim];
    for (int i=0; i<dim; i++) gridSize[i] = dataSize[i] / blockSide;
    
    MDArrayHelper<char> threadH(0, dim, blockSize);
    MDArrayHelper<char> blockH(0, dim, gridSize);
    
    int *threadIndex = new int[dim];
    threadH.getCoords(threadIndex, threadIdx.x);

    int *blockIndex = new int[dim];
    blockH.getCoords(blockIndex, blockIdx.x);

    // initilize helpers for data
    MDArrayHelper<float> inH(in, dim, arrSize);
    MDArrayHelper<float> outH(out, dim, arrSize);
    MDArrayHelper<float> wArrH(wArr, dim, wArrSize);

    // reposition helpers
    int *newPosition = new int[dim];
    for (int i=0; i<dim; i++) newPosition[i] = blockIndex[i] * blockSize[i] + radius[i];

    inH.reposition(newPosition);
    outH.reposition(newPosition);
    wArrH.reposition(radius);

    delete[] newPosition;
    
    // calculate output
    int wArrLinSize = 1;
    for (int i=0; i<dim; i++) wArrLinSize *= wArrSize[i];

    int *wIndex = new int[dim];
    int *currIndex = new int[dim];

    float result = 0;
    for (int i=0; i<wArrLinSize; i++) 
    {
        wArrH.getCoords(wIndex, i);
        for (int j=0; j<dim; j++) currIndex[j] = wIndex[j] + threadIndex[j];

        result += wArrH.get(wIndex) * inH.get(currIndex);
        
    }

    delete[] wIndex;
    delete[] currIndex;

    // write output
    outH.set(result, threadIndex);
  
    // deallocations
    delete[] blockSize;
    delete[] radius;
    delete[] dataSize;
    delete[] gridSize;
    delete[] threadIndex;
    delete[] blockIndex;
}


void print2D(float *arr, int *size)
{
    for (int i=0; i<size[0]; i++) 
    {
        for (int j=0; j<size[1]; j++) cout << arr[i * size[0] + j] << " ";
        cout << endl;
    }
}

#define MAX_N_THREAD 32

/*
in, out: input and output arrays including boundary of radius (wArrSize / 2) at both sides
arrSize: input and output array sizes for each dimention
wArr: weight array
wArrSize: weight array size for each dimention
*/
void applyStencil(float *in, float *out, int *arrSize, float *wArr, int *wArrSize, int dim)
{
    // calculate number of thread per block
    int blockSide = floor(pow(MAX_N_THREAD, (float) 1 / dim));
    int nThread = pow(blockSide, dim);
    
    // create size related variables
    int *radius = (int *) alloca(dim);
    for(int i=0; i<dim; i++) radius[i] = wArrSize[i] / 2;

    int *dataSize = (int *) alloca(dim); // without boundary
    for(int i=0; i<dim; i++) dataSize[i] = arrSize[i] - 2 * radius[i];

    int *extArrSize = (int *) alloca(dim);
    for (int i=0; i<dim; i++)
    {
        int rest = dataSize[i] % blockSide;
        extArrSize[i] = (rest == 0 ? arrSize[i] : arrSize[i] + blockSide - rest);
    }

    int arrLinSize = 1;
    int extArrLinSize = 1;
    int wArrLinSize = 1;
    for (int i=0; i<dim; i++) 
    {
        arrLinSize *= arrSize[i];
        extArrLinSize *= extArrSize[i];
        wArrLinSize *= wArrSize[i];
    }

    // create extended input array
    float *extIn = new float[extArrLinSize];

    MDArrayHelper<float> extInH(extIn, dim, extArrSize);
    MDArrayHelper<float> inH(in, dim, arrSize);

    int *i = (int *) alloca(dim);
    int *start = (int *) alloca(dim);
    int *end = (int *) alloca(dim);

    for (int i=0; i<dim; i++) 
    {
        start[i] = 0;
        end[i] = arrSize[i];
    }

    MDForHost(dim, i, start, end, [&] ()
    {
        extInH.set(inH.get(i), i);
    });
    
    // declare and allocate device arrays
    float *d_in, *d_out, *d_wArr;
    int *d_arrSize, *d_wArrSize;
    
    hipMalloc(&d_in, extArrLinSize * sizeof(float));
    hipMalloc(&d_out, extArrLinSize * sizeof(float));
    hipMalloc(&d_wArr, wArrLinSize * sizeof(float));
    hipMalloc(&d_arrSize, dim * sizeof(int));
    hipMalloc(&d_wArrSize, dim * sizeof(int));

    // copy initial data from host to device
    hipMemcpy(d_in, extIn, extArrLinSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wArr, wArr, wArrLinSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_arrSize, extArrSize, dim * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wArrSize, wArrSize, dim * sizeof(int), hipMemcpyHostToDevice);

    // apply CUDA stencil kernel	
    int *blockNPerDim = (int *) alloca(dim);
    for (int i=0; i<dim; i++) blockNPerDim[i] = (extArrSize[i] - 2 * radius[i]) / blockSide;
    
    int nBlock = 1;
    for (int i=0; i<dim; i++) nBlock *= blockNPerDim[i];
    
	stencilKernel <<< nBlock, nThread >>> (d_in, d_out, d_arrSize, d_wArr, d_wArrSize, dim, blockSide);
    
    // copy output data from device to host
    float *extOut = new float[extArrLinSize];
    hipMemcpy(extOut, d_out, extArrLinSize * sizeof(float), hipMemcpyDeviceToHost);

    // create out from extOut and copy boundaries unchanged
    MDArrayHelper<float> outH(out, dim, arrSize);
    MDArrayHelper<float> extOutH(extOut, dim, extArrSize);

    for (int i=0; i<dim; i++) 
    {
        start[i] = 0;
        end[i] = arrSize[i];
    }

    MDForHost(dim, i, start, end, [&] ()
    {
        bool pred = false; // boundary: true
        for (int a=0; a<dim; a++) if (i[a] < radius[a] || i[a] >= radius[a] + dataSize[a]) pred = true;

        if (pred) outH.set(inH.get(i), i);
        else outH.set(extOutH.get(i), i);
    });
	
	// deallocate device arrays
	hipFree(d_in);
    hipFree(d_out);
    hipFree(d_wArr);
    hipFree(d_arrSize);
    hipFree(d_wArrSize);

    delete[] extIn;
    delete[] extOut;
}



#define ARRSIDE 12
#define WARRSIDE 3

void test2D()
{
    // declare and allocate input, output, and weight arrays
    int dim = 2;
    int dataSize[] = {ARRSIDE, ARRSIDE};
    int wArrSize[] = {WARRSIDE, WARRSIDE};
    
    int *radius = (int *) alloca(dim);
    for(int i=0; i<dim; i++) radius[i] = wArrSize[i] / 2;

    int *arrSize = (int *) alloca(dim);
    for (int i=0; i<dim; i++) arrSize[i] = dataSize[i] + 2 * radius[i];


    int arrLinSize = 1;
    int wArrLinSize = 1;
    for (int i=0; i<dim; i++) 
    {
        arrLinSize *= arrSize[i];
        wArrLinSize *= wArrSize[i];
    }

    float *in = new float[arrLinSize];
    float *out = new float[arrLinSize];
    float *wArr = new float[wArrLinSize];

    // initialize helpers
    MDArrayHelper<float> inHelper(in, dim, arrSize);
    MDArrayHelper<float> outHelper(out, dim, arrSize);
    MDArrayHelper<float> wHelper(wArr, dim, wArrSize);

    // reposision in and out helpers
    inHelper.reposition(radius);
    outHelper.reposition(radius);

    // initialize input array
    int *index = (int *) alloca(dim);
    int *start = (int *) alloca(dim);
    int *end = (int *) alloca(dim);

    for (int i=0; i<dim; i++)
    {
        start[i] = -radius[i];
        end[i] = dataSize[i] + radius[i];
    }

    MDForHost(dim, index, start, end, [&] () 
    {
        bool pred = true; // data: true, boundary: false
        for (int i=0; i<dim; i++) if (index[i] < 0 || index[i] >= dataSize[i]) pred = false;

        if (pred)
        {   // data
            int totIndex = 0;
            for (int i=0; i<dim; i++) totIndex += index[i];
            //inHelper.set((totIndex % 2) + 1, index);
            inHelper.set(1, index);
        }
        else
        {   //boundary
            inHelper.set(5, index);
        }
    });

    cout << "Input: " << endl;
    print2D(in, arrSize);
    cout << endl;

	// initialize output
    for (int linI=0; linI<arrLinSize; linI++) out[linI] = 0;
	
    // initialize weight array
    for (int i=0; i<dim; i++)
    {
        start[i] = 0;
        end[i] = wArrSize[i];
    }

    MDForHost(dim, index, start, end, [&] ()
    {
        wHelper.set((float) 1, index);
    });

    cout << "Weight Array: " << endl;
    print2D(wArr, wArrSize);
    cout << endl;
    
	// apply stencil
	applyStencil(in, out, arrSize, wArr, wArrSize, dim);

	cout << "Output: " << endl;
    print2D(out, arrSize);
    cout << endl;

    // deallocate arrays
    delete[] in;
    delete[] out;
    delete[] wArr;
}

/*
int main()
{
    test2D();
    return 0;
}
*/
