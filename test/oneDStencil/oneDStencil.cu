
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>



__global__
void stencilKernel (int arrSize, float *in, float *out, int wArrSize, float *wArr)
{
    int midIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int radius = wArrSize / 2;
    
    int result = 0;
    for (int i = -1 * radius; i <= radius; i++)
    {
        int arrIndex = midIndex + i;
        if (arrIndex >= 0 && arrIndex < arrSize) result += wArr[i + radius] * in[arrIndex];
    }

    if (midIndex >= 0 && midIndex < arrSize) out[midIndex] = result;
    
    //if (midIndex >= 0 && midIndex < arrSize) out[midIndex] = radius;
    //if (midIndex >= 0 && midIndex < arrSize) out[midIndex] = in[midIndex];
}



int main()
{
    int arrSize = 10000;
    int wArrSize = 5;

    float *in = (float *) malloc(arrSize * sizeof(float));
    float *out = (float *) malloc(arrSize * sizeof(float));
    float *wArr = (float *) malloc(wArrSize * sizeof(float));

    for (int i=0; i<arrSize; i++) in[i] = i % 2;
    for (int i=0; i<wArrSize; i++) wArr[i] = (float) 1 / wArrSize;
    

    float *d_in, *d_out, *d_wArr;
    hipMalloc(&d_in, arrSize * sizeof(float));
    hipMalloc(&d_out, arrSize * sizeof(float));
    hipMalloc(&d_wArr, wArrSize * sizeof(float));

    hipMemcpy(d_in, in, arrSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wArr, wArr, wArrSize * sizeof(float), hipMemcpyHostToDevice);


    int nThread = 128;
    stencilKernel <<< (arrSize + nThread - 1) / nThread, nThread >>> (arrSize, d_in, d_out, wArrSize, d_wArr);


    hipMemcpy(out, d_out, arrSize * sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<10; i++)
    {
        std::cout << out[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_wArr);
    free(in);
    free(out);
    free(wArr);

    return 0;
}

