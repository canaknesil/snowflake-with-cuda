
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>



/*
GPU kernel to perform 1 dim stencil 
on a data including boundary data
using shared memory

in: device array for input data including boundary
out: device array for output data including boundary unchanged
arraySize: size of in and out
wArr: weight array
wArrSize: size of wArr

NOTE: Size of the data part of in and out (without boundaries) is a multiple of block dimention (number of thread per block)
*/
__global__
void stencilKernelShared (float *in, float *out, int arrSize, float *wArr, int wArrSize)
{
	// create index related local variables
    int midIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int radius = wArrSize / 2;
	
	// reposition input and output array pointers for simplicity
	in += radius;
	out += radius;

    // Arrange shared memory
    extern __shared__ float sharedMem[];

    float *sh_in = sharedMem;
	float *sh_wArr = &sh_in[blockDim.x + 2 * radius];
	
	// reposition sh_in array pointer
	sh_in += radius;

    // cache required part of the input array to shared memory
    sh_in[threadIdx.x] = in[midIndex]; // middle
    if (threadIdx.x < radius) sh_in[threadIdx.x - radius] = in[midIndex - radius]; // left
    if (threadIdx.x >= blockDim.x - radius) sh_in[threadIdx.x + radius] = in[midIndex + radius]; // right
    
    // copy boundaries unchanged to out
    //if (midIndex < radius) out[midIndex - radius] = sh_in[threadIdx.x - radius]; // left
    //if (midIndex >= blockDim.x - radius) out[midIndex + radius] = sh_in[threadIdx.x + radius]; // right

    // cache weight array to shared memory if nescessary
    float *wArrPtr;
    if (blockDim.x - 2 * radius >= wArrSize)
    {
        int startId = radius;
        if (threadIdx.x >= startId && threadIdx.x < startId + wArrSize) 
                sh_wArr[threadIdx.x - startId] = wArr[threadIdx.x - startId];
        wArrPtr = sh_wArr;
    }
    else
    {
        wArrPtr = wArr;
    }

    // reposition wArrPtr array pointer
    wArrPtr += radius;
    
    // synchronize threads before starting to access shared memory objects
    __syncthreads();
    
    // calculate output
    float result = 0;
    for (int i = -1 * radius; i <= radius; i++)
    {
        result += wArrPtr[i] * sh_in[threadIdx.x + i];
    }
    
    // write output
    out[midIndex] = result;
}

/*
in, out: input and output arrays including boundary of radius (wArrSize / 2) at both sides
arrSize: input and output array sizes
wArr: weight array
wArrSize: weight array size
*/
void applyStencil(float *in, float *out, int arrSize, float *wArr, int wArrSize)
{
    int nThread = 128; // number of thread per block

    int radius = wArrSize / 2;
    int dataSize = arrSize - 2 * radius; // without boundary
    int extArrSize; // extented array size where dataSize is a multiple of nThread
    {
        int rest = dataSize % nThread;
        extArrSize = (rest == 0 ? arrSize : arrSize + nThread - rest);
    }

	// declare and allocate device arrays
	float *d_in, *d_out, *d_wArr;

    hipMalloc(&d_in, extArrSize * sizeof(float));
    hipMalloc(&d_out, extArrSize * sizeof(float));
    hipMalloc(&d_wArr, wArrSize * sizeof(float));

	// copy initial data from host to device
    hipMemcpy(d_in, in, arrSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_wArr, wArr, wArrSize * sizeof(float), hipMemcpyHostToDevice);

	// apply CUDA stencil kernel
	int sharedMemSize = (wArrSize + nThread + 2 * radius) * sizeof(float);
	
	stencilKernelShared <<< (extArrSize - 2 * radius) / nThread, nThread, sharedMemSize >>> 
			(d_in, d_out, extArrSize, d_wArr, wArrSize);

	// copy output data from device to host
    hipMemcpy(out, d_out, arrSize * sizeof(float), hipMemcpyDeviceToHost);
    
    // copy boundaries unchanged to out
    for (int i=0; i<radius; i++) 
    {
        out[i] = in[i];
        out[i + radius + dataSize] = in[i + radius + dataSize];
    }
	
	// deallocate device arrays
	hipFree(d_in);
    hipFree(d_out);
    hipFree(d_wArr);
}


int main()
{
	// declare and allocate input, output, and weight arrays
    int dataSize = 1000000;
    int wArrSize = 15;
    
	int radius = wArrSize / 2;
	int arrSize = dataSize + 2 * radius;

    float *in = (float *) malloc(arrSize * sizeof(float));
    float *out = (float *) malloc(arrSize * sizeof(float));
    float *wArr = (float *) malloc(wArrSize * sizeof(float));

	// initialize input
	for (int i=0; i<dataSize; i++) in[i + radius] = (i % 2) + 1; // data
	for (int i=0; i<radius; i++) in[i] = in[i + dataSize + radius] = 0; // boundary

	// initialize output
	for (int i=0; i<arrSize; i++) out[i] = 0;
	
	// initialize weight array
    for (int i=0; i<wArrSize; i++) wArr[i] = (float) 1 / wArrSize;
    
	// apply stencil
	applyStencil(in, out, arrSize, wArr, wArrSize);

	// display a portion of output
    for (int i=0; i<20; i++) std::cout << i << ": " << out[i] << std::endl;
    for (int i=arrSize-20; i<arrSize; i++) std::cout << i << ": " << out[i] << std::endl;

    // deallocate arrays
    free(in);
    free(out);
    free(wArr);

    return 0;
}

