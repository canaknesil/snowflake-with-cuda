#include "hip/hip_runtime.h"

#include "MDStencil.h"



__global__
void binOpKernel(int op, float *output, float *left, float *right, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	switch(op)
	{
		case ADD:
			output[i] = left[i] + right[i];
			break;
			
		case SUB:
			output[i] = left[i] - right[i];
			break;
			
		case MUL:
			output[i] = left[i] * right[i];
			break;
			
		case DIV:
			output[i] = left[i] / right[i];
			break;
	}
}


#define THREAD_N 128

void performOp(int op, float *output, float *left, float *right, int size)
{
	int blockN = (size + THREAD_N - 1) / size;
	int extSize = blockN * THREAD_N;
	
	float *d_out, *d_left, *d_right;
	
	hipMalloc(&d_out, extSize * sizeof(float));
	hipMalloc(&d_left, extSize * sizeof(float));
	hipMalloc(&d_right, extSize * sizeof(float));
	
	hipMemcpy(d_left, left, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_right, right, size * sizeof(float), hipMemcpyHostToDevice);
	
	binOpKernel <<< blockN, THREAD_N >>> (op, d_out, d_left, d_right, extSize);
	
	hipMemcpy(output, d_out, size * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_out);
	hipFree(d_left);
	hipFree(d_right);
}